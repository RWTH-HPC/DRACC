/*
Matrix Vector multiplication without allocating the matrix on the device.
*/

#include <hip/hip_runtime.h>
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>

//Grid dimension
#define B 100

//Block dimension
#define T 256

//Array size
#define C B*T

// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                              \
do{                                                                                       \
    hipError_t cuErr = call;                                                             \
    if(hipSuccess != cuErr){                                                             \
      printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr));\
      exit(0);                                                                            \
    }                                                                                     \
}while(0)

//Host pointer for matrix b, input vector a and result vector c
int *a;
int *b;
int *c;

//Device pointer for matrix d_b, input vector d_a and result vector d_c
int *d_a;
int *d_b;
int *d_c;

//Initialization and allocation of the host variables
int init(){
    //Allocating host variables
    a = (int *) malloc(C*sizeof(int));
    b = (int *) malloc(C*C*sizeof(int));
    c = (int *) malloc(C*sizeof(int));
    
    //Initialize host values
    for(int i=0; i<C; i++){
        for(int j=0; j<C; j++){
            b[j+i*C]=1;
        }
        a[i]=1;
        c[i]=0;
    }
        return 0;
}

//Kernel
__global__ void Mult(int* d_a, int* d_b, int* d_c){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    
    for(int j=0; j<C; j++){
        d_c[tid]+=d_b[j+tid*C]*d_a[j];
    }
}

//Checking if the values stored in c are correct
int check(){
    bool test = false;
    for(int i=0; i<C; i++){
        if(c[i]!=C){
            test = true;
        }
    }
    printf("Data Race occured: %s\n",test ? "true\n" : "false\n");
    return 0;
    
    
}

//Initialization of the variables on the GPU 
int initcuda(){
    //Allocation of GPU memory for d_a,d_c
    cudaErrorCheck( hipMalloc(&d_a, C*sizeof(int)));
    //cudaErrorCheck( cudaMalloc(&d_b, C*C*sizeof(int)));
    cudaErrorCheck( hipMalloc(&d_c, C*sizeof(int)));
    
    //Copying the array a and the matrix b from the host to the array d_a and the matrix d_b on the device
    cudaErrorCheck( hipMemcpy(d_a,a,C*sizeof(int),hipMemcpyHostToDevice));
    cudaErrorCheck( hipMemcpy(d_b,b,C*C*sizeof(int),hipMemcpyHostToDevice));
    
    return 0;
}

//Main programm
int main(){
    //Calling the initialization methods
    init();
    initcuda();
    
    //Launch Kernel
    Mult<<<B,T>>>(d_a,d_b,d_c);
    
    // Check for errors in kernel launch (e.g. invalid execution configuration paramters)
    cudaErrorCheck( hipGetLastError());

    // Check for errors on the GPU after control is returned to CPU
    cudaErrorCheck( hipDeviceSynchronize());
    
    //Copying back the result d_c from the device to the host array c
    cudaErrorCheck( hipMemcpy(c,d_c,C*sizeof(int),hipMemcpyDeviceToHost)); 
    
    //Verify result
    check();
    
    //Freeing GPU memory
    cudaErrorCheck( hipFree(d_a));
    cudaErrorCheck( hipFree(d_b));
    cudaErrorCheck( hipFree(d_c));
    
    //Freeing CPU memory
    free(a);
    free(b);
    free(c);
    return 0;
}