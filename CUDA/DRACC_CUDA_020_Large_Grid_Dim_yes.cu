/*
Matrix addition with a too large grid dimension.
*/

#include <hip/hip_runtime.h>
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

// Grid size
#define B 5120

// Block size
#define T 1

// Matrix dimension
#define C B*T

// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                              \
do{                                                                                       \
    hipError_t cuErr = call;                                                             \
    if(hipSuccess != cuErr){                                                             \
      printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr));\
      exit(0);                                                                            \
    }                                                                                     \
}while(0)

int size = sizeof(int);

// Host pointer
int *a;
int *b;
int *c;

// Device pointer
int *d_a;
int *d_b;
int *d_c;

// Host initialisation and matrix allocation
int init(){
    a = (int *) malloc(C*C*size);
    b = (int *) malloc(C*C*size);
    c = (int *) malloc(C*C*size);
    for(int i=0; i<C; i++){
        for(int j=0; j<C; j++){
            b[j+i*C]=1;
            a[j+i*C]=1;
            c[j+i*C]=0;
        }
        
    }
        return 0;
}

// Kernel
__global__ void Mult(int* d_a, int* d_b, int* d_c){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    d_c[tid] = d_a[tid] + d_b[tid];
    
}

// Verifying results
int check(){
    bool test = false;
    for(int i=0; i<C*C; i++){
        if(c[i]!=2){
            test = true;
        }
    }
    printf("Data Race occured: %s\n",test ? "true\n" : "false\n");
    return 0;
    
    
}

// Allocating device memory and copying matrices a and b from the host to d_a and d_b on the device
void initcuda(){
    
    cudaErrorCheck( hipMalloc(&d_a, C*C*size));
    cudaErrorCheck( hipMalloc(&d_b, C*C*size));
    cudaErrorCheck( hipMalloc(&d_c, C*C*size));
    cudaErrorCheck( hipMemcpy(d_a,a,C*C*size,hipMemcpyHostToDevice));
    cudaErrorCheck( hipMemcpy(d_b,b,C*C*size,hipMemcpyHostToDevice));
    
}

// Main program
int main(){
    // Initialisation
    init();
    initcuda();
    
    //Launch Kernel
    Mult<<<B,T>>>(d_a,d_b,d_c);
    
     // Check for errors in kernel launch (e.g. invalid execution configuration paramters)
    cudaErrorCheck( hipGetLastError());

    // Check for errors on the GPU after control is returned to CPU
    cudaErrorCheck( hipDeviceSynchronize());
    
    // Copying back the result d_c from the device to c on the host
    cudaErrorCheck( hipMemcpy(c,d_c,C*C*size,hipMemcpyHostToDevice));
    
    // Verifying results
    check();
    
    // Freeing device memory
    cudaErrorCheck( hipFree(d_a));
    cudaErrorCheck( hipFree(d_b));
    cudaErrorCheck( hipFree(d_c));
    
    // Freeing host memory
    free(a);
    free(b);
    free(c);
    return 0;
}