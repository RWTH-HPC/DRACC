/*
Concurrent access on a counter with no lock. Atomicity Violation. Data Race in line 24. Inter and Intra Region.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                              \
do{                                                                                       \
    hipError_t cuErr = call;                                                             \
    if(hipSuccess != cuErr){                                                             \
      printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr));\
      exit(0);                                                                            \
    }                                                                                     \
}while(0)

// Grid dimension
#define B 100

// Block dimension
#define T 512

// Kernel
__global__ void count(int *d_countervar){
    d_countervar[0]++;
}

// Main program 
int main(){

    // Device pointer for counter variable
    int *d_count;
    
    // Allocation of host counter variable
    int *countervar = (int *) malloc(sizeof(int));
    // Initialization of the counter variable
    countervar[0] = 0;
    
    // Allocation of GPU memory
    cudaErrorCheck( hipMalloc(&d_count, sizeof(int)));
    
    // Copying the counter variable from the host to the device
    cudaErrorCheck( hipMemcpy(d_count,countervar,sizeof(int),hipMemcpyHostToDevice));
    
    //Launch Kernel
    count<<<B,T>>>(d_count);
    
     // Check for errors in kernel launch (e.g. invalid execution configuration paramters)
    cudaErrorCheck( hipGetLastError());

    // Check for errors on the GPU after control is returned to CPU
    cudaErrorCheck( hipDeviceSynchronize());
    
    // Copying the counter variable from the device to the host
    cudaErrorCheck( hipMemcpy(countervar,d_count,sizeof(int),hipMemcpyDeviceToHost));
    
    // Verifying result
    printf("counter: %i expected: %i \n ", countervar[0], T*B);
    
    // Freeing GPU memory
    cudaErrorCheck( hipFree(d_count));
    
    // Freeing CPU memory
    free(countervar);
    return 0;
}