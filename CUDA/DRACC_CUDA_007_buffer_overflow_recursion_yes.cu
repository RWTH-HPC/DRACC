/*
Recursive call of count to force a buffer overflow due to overhead.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                              \
do{                                                                                       \
    hipError_t cuErr = call;                                                             \
    if(hipSuccess != cuErr){                                                             \
      printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr));\
      exit(0);                                                                            \
    }                                                                                     \
}while(0)

// Grid dimension
#define B 100

// Block dimension
#define T 512


// Device recursion
__device__ void recurs(int *d_countervar){
//    int tmp[T];
//    for(int i=0;i<T;i++){
//        tmp[i]=d_countervar[0];
//    }
    atomicAdd(d_countervar,1);
//    if(d_countervar[0]<5120000){
    recurs(d_countervar);
//    }
//    atomicAdd(d_countervar,temp);
    
}

// Kernel
__global__ void count( int *d_countervar){
    recurs(d_countervar);
}

// Main program 
int main(){

    // Device pointer for counter variable
    int *d_count;
    
    // Allocation of host counter variable
    int *countervar = (int *) malloc(sizeof(int));
    // Initialization of the counter variable
    countervar[0] = 0;
    
    // Allocation of GPU memory
    cudaErrorCheck( hipMalloc(&d_count, sizeof(int)));
    
    // Copying the counter variable from the host to the device
    cudaErrorCheck( hipMemcpy(d_count,countervar,sizeof(int),hipMemcpyHostToDevice));
    
    // Stack limit 
    cudaErrorCheck( hipDeviceSetLimit(hipLimitStackSize,1000));
    
    //Launch Kernel
    count<<<B,T>>>(d_count);
    
    // Check for errors in kernel launch (e.g. invalid execution configuration paramters)
    cudaErrorCheck( hipGetLastError());

    // Check for errors on the GPU after control is returned to CPU
    cudaErrorCheck( hipDeviceSynchronize());
    
    // Copying the counter variable from the device to the host
    cudaErrorCheck( hipMemcpy(countervar,d_count,sizeof(int),hipMemcpyDeviceToHost));
    
    // Verifying result
    printf("counter: %i expected: %i \n ", countervar, T*B);
    
    // Freeing GPU memory
    cudaErrorCheck( hipFree(d_count));
    
    // Freeing CPU memory
    free(countervar);
    return 0;
}