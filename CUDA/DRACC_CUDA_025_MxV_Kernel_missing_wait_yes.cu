/*
Matrix Vector multiplication without a barrier at the end of execution to wait for the tasks to finish, before evaluating the results.
*/

#include <hip/hip_runtime.h>
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>

//Grid dimension
#define B 100

//Block dimension
#define T 256

//Array size
#define C B*T

// Iteration number
#define N 100

// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                              \
do{                                                                                       \
    hipError_t cuErr = call;                                                             \
    if(hipSuccess != cuErr){                                                             \
      printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr));\
      exit(0);                                                                            \
    }                                                                                     \
}while(0)

//cuda Streams
hipStream_t stream1, stream2;

//Host pointer for matrix b, input vector a and result vector c
int *a;
int *b;
int *c;

//Device pointer for matrix d_b, input vector d_a and result vector d_c
int *d_a;
int *d_b;
int *d_c;

//Initialization and allocation of the host variables
int init(){
    //Allocating host variables
    a = (int *) malloc(C*sizeof(int));
    b = (int *) malloc(C*C*sizeof(int));
    //c = (int *) malloc(C*sizeof(int));
	cudaErrorCheck(hipHostMalloc(&c, C*sizeof(int), hipHostMallocDefault));
    
    //Initialize host values
    for(int i=0; i<C; i++){
        for(int j=0; j<C; j++){
            b[j+i*C]=1;
        }
        a[i]=1;
        c[i]=0;
    }
        return 0;
}

//Kernel
__global__ void Mult(int* d_a, int* d_b, int* d_c){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    
    for(int i=0; i<N; i++){
        d_c[tid] = 0;
        for(int j=0; j<C; j++){
            d_c[tid]+=d_b[j+tid*C]*d_a[j];
        }
        
    }
}

//Checking if the values stored in c are correct
int check(){
    bool test = false;
    for(int i=C-1; i>=0; i--){
        //printf("c[%i]:%i\n",i,c[i]);
        if(c[i]!=C){
            test = true;
        }
    }
    printf("Data Race occured: %s\n",test ? "true\n" : "false\n");
    return 0;
    
    
}

//Initialization of the variables on the GPU 
int initcuda(){
	//create cuda streams
	cudaErrorCheck(hipStreamCreate ( &stream1) );
	cudaErrorCheck(hipStreamCreate ( &stream2) );
	
    //Allocation of GPU memory for d_a,d_b,d_c
    cudaErrorCheck( hipMalloc(&d_a, C*sizeof(int)));
    cudaErrorCheck( hipMalloc(&d_b, C*C*sizeof(int)));
    cudaErrorCheck( hipMalloc(&d_c, C*sizeof(int)));
    
    //Copying the array a and the matrix b from the host to the array d_a and the matrix d_b on the device
    cudaErrorCheck( hipMemcpy(d_a,a,C*sizeof(int),hipMemcpyHostToDevice));
    cudaErrorCheck( hipMemcpy(d_b,b,C*C*sizeof(int),hipMemcpyHostToDevice));
    
    return 0;
}

//Main programm
int main(){
    //Calling the initialization methods
    init();
    initcuda();
    
    //Launch Kernel
    Mult<<<B,T,0,stream1>>>(d_a,d_b,d_c);
	   
    // Check for errors in kernel launch (e.g. invalid execution configuration paramters)
    cudaErrorCheck( hipGetLastError());
   
    //Copying back half the result d_c from the device to the host array c
    cudaErrorCheck(hipMemcpyAsync(c,d_c,C*sizeof(int),hipMemcpyDeviceToHost,stream2)); 

	// Check for errors on the GPU after control is returned to CPU
    cudaErrorCheck( hipDeviceSynchronize());
 
	//Verify result
    check();    
	
    //Freeing GPU memory
    cudaErrorCheck( hipFree(d_a));
    cudaErrorCheck( hipFree(d_b));
    cudaErrorCheck( hipFree(d_c));
    
    //Freeing CPU memory
    free(a);
    free(b);
    //free(c);
	cudaErrorCheck(hipHostFree(c));
    return 0;
}