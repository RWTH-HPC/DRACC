/*
Concurrent access on an atimoc counter. Intra Region.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                              \
do{                                                                                       \
    hipError_t cuErr = call;                                                             \
    if(hipSuccess != cuErr){                                                             \
      printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr));\
      exit(0);                                                                            \
    }                                                                                     \
}while(0)

// Incrementations per thread
#define B 100

// Block dimension
#define T 512

// Kernel
__global__ void count(int *d_countervar){
    for(int i=0; i<B;i++){
    atomicAdd_block(d_countervar,1);}
}

// Main program 
int main(){

    // Device pointer for counter variable
    int *d_count;
    
    // Allocation of host counter variable
    int *countervar = (int *) malloc(sizeof(int));
    // Initialization of the counter variable
    countervar[0] = 0;
    
    // Allocation of GPU memory
    cudaErrorCheck( hipMalloc(&d_count, sizeof(int)));
    
    // Copying the counter variable from the host to the device
    cudaErrorCheck( hipMemcpy(d_count,countervar,sizeof(int),hipMemcpyHostToDevice));
    
    //Launch Kernel
    count<<<1,T>>>(d_count);
    
    // Check for errors in kernel launch (e.g. invalid execution configuration paramters)
    cudaErrorCheck( hipGetLastError());

    // Check for errors on the GPU after control is returned to CPU
    cudaErrorCheck( hipDeviceSynchronize());
    
    // Copying the counter variable from the device to the host
    cudaErrorCheck( hipMemcpy(countervar,d_count,sizeof(int),hipMemcpyDeviceToHost));
    
    // Verifying result
    printf("counter: %i expected: %i \n ", countervar[0], T*B);
    
    // Freeing GPU memory
    cudaErrorCheck( hipFree(d_count));
    
    // Freeing CPU memory
    free(countervar);
    return 0;
}