/*
Incrementaion with a threshold of N. The incrementation is atomic, but the query on the threshold depends on the counter.
The query is not atomic, therefore a Data Race is possible in line 30 and 31.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                              \
do{                                                                                       \
    hipError_t cuErr = call;                                                             \
    if(hipSuccess != cuErr){                                                             \
      printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr));\
      exit(0);                                                                            \
    }                                                                                     \
}while(0)

// Grid dimension
#define B 128

// Block dimension
#define T 512

// Count iterations
#define N 200000


// Kernel
__global__ void count( int *d_countervar){
    for(int i=0; i<N*2; i++){
        if(d_countervar[0]<N){
            atomicAdd(d_countervar,1);
        }
    }
}

// Main program 
int main(){

    // Device pointer for counter variable
    int *d_count;
    
    // Allocation of host counter variable
    int *countervar = (int *) malloc(sizeof(int));
    // Initialization of the counter variable
    countervar[0] = 0;
    
    // Allocation of GPU memory
    cudaErrorCheck( hipMalloc(&d_count, sizeof(int)));
    
    // Copying the counter variable from the host to the device
    cudaErrorCheck( hipMemcpy(d_count,countervar,sizeof(int),hipMemcpyHostToDevice));
    
    //Launch Kernel
    count<<<B,T>>>(d_count);
    
     // Check for errors in kernel launch (e.g. invalid execution configuration paramters)
    cudaErrorCheck( hipGetLastError());

    // Check for errors on the GPU after control is returned to CPU
    cudaErrorCheck( hipDeviceSynchronize());
    
    // Copying the counter variable from the device to the host
    cudaErrorCheck( hipMemcpy(countervar,d_count,sizeof(int),hipMemcpyDeviceToHost));
    
    // Verifying result
    printf("counter: %i expected: %i \n ", countervar[0], N);
    
    // Freeing GPU memory
    cudaErrorCheck( hipFree(d_count));
    
    // Freeing CPU memory
    free(countervar);
    return 0;
}