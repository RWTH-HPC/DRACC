/*
Array with backwards dependencies. Order Violation. Data Race in line 37. Inter nd Intra Region.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                              \
do{                                                                                       \
    hipError_t cuErr = call;                                                             \
    if(hipSuccess != cuErr){                                                             \
      printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr));\
      exit(0);                                                                            \
    }                                                                                     \
}while(0)

// Grid dimension
#define B 100

// Block dimension
#define T 512

// Host pointer
int *countervar;

// Initialization
int init(){
    for(int i=0; i<B*T; i++){
        countervar[i]=0;
    }
    return 0;
}

// Kernel
__global__ void count(int *countervar){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid!=0){
        countervar[tid] = countervar[tid-1] + 1;
    }
}

// Verifying result

int check(){
    bool test = false;
    for(int i=0; i<B*T; i++){
        if(countervar[i]!=i){
            test = true;
        }
    }
    printf("Data Race occured: %s\n",test ? "true" : "false");
    return 0;
}

// Main program 
int main(){

    // Device pointer for counter variable
    int *d_count;
    
    // Allocation of host counter variable
    countervar = (int *) malloc(B*T*sizeof(int));
    // Initialization of the counter variable
    init();
    
    // Allocation of GPU memory
    cudaErrorCheck( hipMalloc(&d_count, B*T*sizeof(int)));
    
    // Copying the counter variable from the host to the device
    cudaErrorCheck( hipMemcpy(d_count,countervar,B*T*sizeof(int),hipMemcpyHostToDevice));
    
    //Launch Kernel
    count<<<B,T>>>(d_count);
    
     // Check for errors in kernel launch (e.g. invalid execution configuration paramters)
    cudaErrorCheck( hipGetLastError());

    // Check for errors on the GPU after control is returned to CPU
    cudaErrorCheck( hipDeviceSynchronize());
    
    // Copying the counter variable from the device to the host
    cudaErrorCheck( hipMemcpy(countervar,d_count,sizeof(int),hipMemcpyDeviceToHost));
    
    // Verifying result
    check();
    
    // Freeing GPU memory
    cudaErrorCheck( hipFree(d_count));
    
    // Freeing CPU memory
    free(countervar);
    return 0;
}