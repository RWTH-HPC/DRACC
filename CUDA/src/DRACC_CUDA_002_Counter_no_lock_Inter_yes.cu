/*
Concurrent access on a counter with no lock. Atomicity Violation. Data Race in line 26. Inter Region Data Race.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                              \
do{                                                                                       \
    hipError_t cuErr = call;                                                             \
    if(hipSuccess != cuErr){                                                             \
      printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr));\
      exit(0);                                                                            \
    }                                                                                     \
}while(0)

// Grid dimension
#define B 100

// Incrementations per block
#define T 512

// Kernel
__global__ void count(int *d_countervar){
    
    for(int i=0; i<T;i++){
    d_countervar[0] ++;
    }
}

// Main program 
int main(){

    // Device pointer for counter variable
    int *d_count;
    
    // Allocation of host counter variable
    int *countervar = (int *) malloc(sizeof(int));
    // Initialization of the counter variable
    countervar[0] = 0;
    
    // Allocation of GPU memory
    cudaErrorCheck( hipMalloc(&d_count, sizeof(int)));
    
    // Copying the counter variable from the host to the device
    cudaErrorCheck( hipMemcpy(d_count,countervar,sizeof(int),hipMemcpyHostToDevice));
    
    //Launch Kernel
    count<<<B,1>>>(d_count);
    
    // Check for errors in kernel launch (e.g. invalid execution configuration paramters)
    cudaErrorCheck( hipGetLastError());

    // Check for errors on the GPU after control is returned to CPU
    cudaErrorCheck( hipDeviceSynchronize());
    
    // Copying the counter variable from the device to the host
    cudaErrorCheck( hipMemcpy(countervar,d_count,sizeof(int),hipMemcpyDeviceToHost));
    
    // Verifying result
    printf("counter: %i expected: %i \n ", countervar[0], T*B);
    
    // Freeing GPU memory
    cudaErrorCheck( hipFree(d_count));
    
    // Freeing CPU memory
    free(countervar);
    return 0;
}