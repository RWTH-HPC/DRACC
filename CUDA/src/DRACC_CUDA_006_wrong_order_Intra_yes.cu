/*
Array with backwards dependencies. Order Violation. Data Race in line 38. Intra Region Data Race.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                              \
do{                                                                                       \
    hipError_t cuErr = call;                                                             \
    if(hipSuccess != cuErr){                                                             \
      printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr));\
      exit(0);                                                                            \
    }                                                                                     \
}while(0)

// Iterations per thread
#define B 100

// Block dimension
#define T 512

// Host pointer
int *countervar;

// Initialization
int init(){
    for(int i=0; i<B*T; i++){
        countervar[i]=0;
    }
    return 0;
}

// Kernel
__global__ void count(int *countervar){
    int i;
    for(i = threadIdx.x * B; i < threadIdx.x * B + B; i++){
        if(i!=0){
            countervar[i] = countervar[i-1] + 1;
        }
    }
}

// Verifying result
int check(){
    bool test = false;
    for(int i=0; i<B*T; i++){
        if(countervar[i]!=i){
            test = true;
        }
    }
    printf("Memory Access Issue visible: %s\n",test ? "true" : "false");
    return 0;
}

// Main program 
int main(){

    // Device pointer for counter variable
    int *d_count;
    
    // Allocation of host counter variable
    countervar = (int *) malloc(B*T*sizeof(int));
    // Initialization of the counter variable
    init();
    
    // Allocation of GPU memory
    cudaErrorCheck( hipMalloc(&d_count, B*T*sizeof(int)));
    
    // Copying the counter variable from the host to the device
    cudaErrorCheck( hipMemcpy(d_count,countervar,B*T*sizeof(int),hipMemcpyHostToDevice));
    
    //Launch Kernel
    count<<<1,T>>>(d_count);
    
     // Check for errors in kernel launch (e.g. invalid execution configuration paramters)
    cudaErrorCheck( hipGetLastError());

    // Check for errors on the GPU after control is returned to CPU
    cudaErrorCheck( hipDeviceSynchronize());
    
    // Copying the counter variable from the device to the host
    cudaErrorCheck( hipMemcpy(countervar,d_count,B*T*sizeof(int),hipMemcpyDeviceToHost));
    
    // Verifying result
    check();
    
    // Freeing GPU memory
    cudaErrorCheck( hipFree(d_count));
    
    // Freeing CPU memory
    free(countervar);
    return 0;
}
