/*
Matrix Vector multiplication with an update of a variable during asynchroneous execution of the kernel.
*/

#include <hip/hip_runtime.h>
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>

//Grid dimension
#define B 100

//Block dimension
#define T 256

//Array size
#define C B*T

// Iteration number
#define N 100

// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                              \
do{                                                                                       \
    hipError_t cuErr = call;                                                             \
    if(hipSuccess != cuErr){                                                             \
      printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr));\
      exit(0);                                                                            \
    }                                                                                     \
}while(0)

//cuda Streams
hipStream_t stream1, stream2;

//Host pointer for matrix b, input vector a and result vector c
int *a;
int *b;
int *c;

//Device pointer for matrix d_b, input vector d_a and result vector d_c
int *d_a;
int *d_b;
int *d_c;

//Initialization and allocation of the host variables
int init(){
    //Allocating host variables
    //a = (int *) malloc(C*sizeof(int));
    b = (int *) malloc(C*C*sizeof(int));
    c = (int *) malloc(C*sizeof(int));
	cudaErrorCheck(hipHostMalloc(&a, C*sizeof(int), hipHostMallocDefault));
    
    //Initialize host values
    for(int i=0; i<C; i++){
        for(int j=0; j<C; j++){
            b[j+i*C]=1;
        }
        a[i]=1;
        c[i]=0;
    }
        return 0;
}

//Interrupting memory movement
int Update(){
	for(int i=C-1;i>=0;i--){
		a[i] = C;
		cudaErrorCheck(hipMemcpyAsync(d_a,a,C*sizeof(int),hipMemcpyDeviceToHost,stream2)); 
	}
	return 0;
}

//Kernel
__global__ void Mult(int* d_a, int* d_b, int* d_c){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    
    for(int i=0; i<N; i++){
        d_c[tid] = 0;
        for(int j=0; j<C; j++){
            d_c[tid]+=d_b[j+tid*C]*d_a[j];
        }
        
    }
}

//Checking if the values stored in c are correct
int check(){
    bool test = false;
    for(int i=C-1; i>=0; i--){
        //printf("c[%i]:%i\n",i,c[i]);
        if(c[i]!=C){
            test = true;
        }
    }
    printf("Data Race occured: %s\n",test ? "true\n" : "false\n");
    return 0;
    
    
}

//Initialization of the variables on the GPU 
int initcuda(){
	//create cuda streams
	cudaErrorCheck(hipStreamCreate ( &stream1) );
	cudaErrorCheck(hipStreamCreate ( &stream2) );
	
    //Allocation of GPU memory for d_a,d_b,d_c
    cudaErrorCheck( hipMalloc(&d_a, C*sizeof(int)));
    cudaErrorCheck( hipMalloc(&d_b, C*C*sizeof(int)));
    cudaErrorCheck( hipMalloc(&d_c, C*sizeof(int)));
    
    //Copying the array a and the matrix b from the host to the array d_a and the matrix d_b on the device
    cudaErrorCheck( hipMemcpy(d_a,a,C*sizeof(int),hipMemcpyHostToDevice));
    cudaErrorCheck( hipMemcpy(d_b,b,C*C*sizeof(int),hipMemcpyHostToDevice));
    
    return 0;
}

//Main programm
int main(){
    //Calling the initialization methods
    init();
    initcuda();
    
    //Launch Kernel
    Mult<<<B,T,0,stream1>>>(d_a,d_b,d_c);
	   
	//Memcopy to overwright a
	Update();
	   
    // Check for errors in kernel launch (e.g. invalid execution configuration paramters)
    cudaErrorCheck( hipGetLastError());

	// Check for errors on the GPU after control is returned to CPU
    cudaErrorCheck( hipDeviceSynchronize());
	
	//Copying back the result d_c from the device to the host array c
    cudaErrorCheck(hipMemcpy(c,d_c,C*sizeof(int),hipMemcpyDeviceToHost)); 
 
	//Verify result
    check();    
	
    //Freeing GPU memory
    cudaErrorCheck( hipFree(d_a));
    cudaErrorCheck( hipFree(d_b));
    cudaErrorCheck( hipFree(d_c));
    
    //Freeing CPU memory
    //free(a);
    free(b);
    free(c);
	cudaErrorCheck(hipHostFree(a));
    return 0;
}